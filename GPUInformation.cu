#include <bitset>
#include <iomanip>
#include <ios>
#include <iostream>
#include <sstream>
#include <string>

std::string get_UUID_as_String(const hipUUID& uuid){
  std::stringstream result;
  result << "GPU-";
  size_t cnt = 0;
  for(auto& c: uuid.bytes) {
    std::bitset<8> bits(c);
    if(cnt == 4 || cnt == 6 || cnt == 8 || cnt == 10) result << "-";
    result << std::hex << bits.to_ulong() ;
    cnt++;
  }
  return result.str();
}

void print_device_information(const int deviceId) {
  hipDeviceProp_t deviceProp{};
  hipGetDeviceProperties(&deviceProp, deviceId);

  std::cout << "================ DeviceId: " << deviceId << " ================ \n";
  std::cout << "--> General Information: \n"
            << "\tDevice name: " << deviceProp.name << "\n"
            << "\tUUID: " << get_UUID_as_String(deviceProp.uuid) << "\n"
            << "\tIntegrated: " << deviceProp.integrated << "\n"
            << "\tClock rate (kHz): " << deviceProp.clockRate << "\n";

  std::cout << "\n--> Computation: \n"
            << "\tComputer capability: " << deviceProp.major << "." << deviceProp.minor << "\n"
            << "\t# of SMs: " << deviceProp.multiProcessorCount << "\n"
            << "\tWarp size: " << deviceProp.warpSize << "\n"
            << "\tmax block dim: (" << deviceProp.maxThreadsDim[0] << ", " << deviceProp.maxThreadsDim[1] << ", "
            << deviceProp.maxThreadsDim[2] << ")\n"
            << "\tmax threads/block: " << deviceProp.maxThreadsPerBlock << "\n"
            << "\tmax threads/SM: " << deviceProp.maxThreadsPerMultiProcessor << "\n"
            << "\tSingle/Double precision ration: " << deviceProp.singleToDoublePrecisionPerfRatio << "\n"
            << "\n";

  std::cout << "--> Memory: \n"
            << "\tUnified addressing: " << deviceProp.unifiedAddressing << "\n"
            << "\tSupports managed memory: " << deviceProp.managedMemory << "\n"
            << "\tTotal global memory (Gb): " << std::setprecision(3) << std::fixed
            << static_cast<float>(deviceProp.totalGlobalMem) / (1024. * 1024. * 1024.) << "\n"
            << "\tTotal constant memory (kb): " << deviceProp.totalConstMem / 1024 << "\n"
            << "\tsMem/block (kb): " << deviceProp.sharedMemPerBlock / 1024 << "\n"
            << "\tsMem/SM (kb): " << deviceProp.sharedMemPerMultiprocessor << "\n"
            << "\n";
}

int main() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  std::cout << "Detected " << deviceCount << " GPU devices.\n";

  for (int device = 0; device < deviceCount; ++device) {
    print_device_information(device);
  }

  return 0;
}
